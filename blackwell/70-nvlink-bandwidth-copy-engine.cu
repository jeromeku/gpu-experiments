#include "hip/hip_runtime.h"
/*
    Transfer time: 13.772 ms
    Bandwidth: 726.13 GB/s
*/

#include "kittens.cuh"

// Kernel to initialize memory with a value
__global__ void initKernel(float* data, float value, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
    for (size_t i = idx; i < n; i += stride) {
        data[i] = value;
    }
}

// Kernel to verify data correctness
__global__ void verifyKernel(float* data, float expected, size_t n, int* errorCount) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
    for (size_t i = idx; i < n; i += stride) {
        if (fabsf(data[i] - expected) > 1e-5f) {
            atomicAdd(errorCount, 1);
        }
    }
}

int main() {
    // Configuration
    const size_t dataSize = 10ULL * 1024 * 1024 * 1024;  // 10 GB
    const size_t numElements = dataSize / sizeof(float);
    const float srcValue = 3.14f;
    const float dstInitValue = 0.0f;
    
    printf("NVLink Unidirectional Bandwidth Test\n");
    printf("=====================================\n");
    printf("Data size: %.2f GB\n", dataSize / (1024.0 * 1024.0 * 1024.0));
    printf("Number of float elements: %zu\n", numElements);
    
    // Allocate memory on device 0
    float* d0_data;
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipMalloc(&d0_data, dataSize));
    printf("Allocated %.2f GB on Device 0\n", dataSize / (1024.0 * 1024.0 * 1024.0));

    // Initialize device 0 memory with 3.14
    int blockSize = 256;
    int gridSize = (numElements + blockSize - 1) / blockSize;
    // Limit grid size to avoid excessive blocks
    gridSize = min(gridSize, 65536);

    initKernel<<<gridSize, blockSize>>>(d0_data, srcValue, numElements);
    CUDACHECK(hipDeviceSynchronize());
    printf("Initialized Device 0 memory with value: %.2f\n", srcValue);
    
    // Allocate memory on device 1
    float* d1_data;
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMalloc(&d1_data, dataSize));
    printf("Allocated %.2f GB on Device 1\n", dataSize / (1024.0 * 1024.0 * 1024.0));
    
    // Initialize device 1 memory with 0
    initKernel<<<gridSize, blockSize>>>(d1_data, dstInitValue, numElements);
    CUDACHECK(hipDeviceSynchronize());
    printf("Initialized Device 1 memory with value: %.2f\n\n", dstInitValue);
    
    // Create events on device 0 for timing
    CUDACHECK(hipSetDevice(0));
    hipStream_t stream;
    hipEvent_t start, stop;
    CUDACHECK(hipStreamCreate(&stream));
    CUDACHECK(hipEventCreate(&start));
    CUDACHECK(hipEventCreate(&stop));
    
    // Enable peer access if not already enabled
    CUDACHECK(hipDeviceEnablePeerAccess(1, 0));
    
    // Warm up run
    printf("\nPerforming warm-up transfer...\n");
    CUDACHECK(hipMemcpyPeerAsync(d1_data, 1, d0_data, 0, dataSize, stream));
    CUDACHECK(hipStreamSynchronize(stream));
    
    // Timed transfer: Device 0 -> Device 1
    printf("\nStarting timed transfer: Device 0 -> Device 1\n");
    
    // Record start event
    CUDACHECK(hipEventRecord(start, stream));
    
    // Perform async memory copy
    CUDACHECK(hipMemcpyPeerAsync(d1_data, 1, d0_data, 0, dataSize, stream));
    
    // Record stop event
    CUDACHECK(hipEventRecord(stop, stream));
    
    // Wait for transfer to complete
    CUDACHECK(hipStreamSynchronize(stream));
    
    // Calculate elapsed time
    float milliseconds = 0;
    CUDACHECK(hipEventElapsedTime(&milliseconds, start, stop));
    
    // Calculate bandwidth
    double seconds = milliseconds / 1000.0;
    double gigabytes = dataSize / (1024.0 * 1024.0 * 1024.0);
    double bandwidth_GBps = gigabytes / seconds;
    
    printf("\nTransfer Results:\n");
    printf("-----------------\n");
    printf("Transfer time: %.3f ms\n", milliseconds);
    printf("Bandwidth: %.2f GB/s\n", bandwidth_GBps);
    
    // Verify correctness on device 1
    printf("\nVerifying data correctness on Device 1...\n");
    CUDACHECK(hipSetDevice(1));
    
    int* d_errorCount;
    CUDACHECK(hipMalloc(&d_errorCount, sizeof(int)));
    CUDACHECK(hipMemset(d_errorCount, 0, sizeof(int)));
    
    verifyKernel<<<gridSize, blockSize>>>(d1_data, srcValue, numElements, d_errorCount);
    CUDACHECK(hipDeviceSynchronize());
    
    int h_errorCount;
    CUDACHECK(hipMemcpy(&h_errorCount, d_errorCount, sizeof(int), hipMemcpyDeviceToHost));
    
    if (h_errorCount == 0) {
        printf("✓ Correctness check PASSED: All values match expected value (%.2f)\n", srcValue);
    } else {
        printf("✗ Correctness check FAILED: %d mismatches found\n", h_errorCount);
    }
    
    // Sample a few values for verification
    float sample[10];
    CUDACHECK(hipMemcpy(sample, d1_data, sizeof(sample), hipMemcpyDeviceToHost));
    printf("\nFirst 10 values on Device 1 after transfer: ");
    for (int i = 0; i < 10; i++) {
        printf("%.2f ", sample[i]);
    }
    printf("\n");
    
    // Cleanup
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipFree(d0_data));
    CUDACHECK(hipEventDestroy(start));
    CUDACHECK(hipEventDestroy(stop));
    
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipFree(d1_data));
    CUDACHECK(hipFree(d_errorCount));
    
    printf("\nTest completed successfully!\n");
    
    return 0;
}
